#include "hip/hip_runtime.h"
// #include <math.h>
#include <cmath>
#include <iostream>
#include "hip/hip_runtime.h"
// #include <stdlib.h>
#include <vector>
// #include <time.h>
#include "dev_array.h"

using namespace std;

__global__ void matrixMultiplicationKernel(double* A, double* B, double* C, int N) {

    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    double tmpSum = 0;

    if (ROW < N && COL < N) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < N; i++) {
            tmpSum += A[ROW * N + i] * B[i * N + COL];
        }
    }
    C[ROW * N + COL] = tmpSum;
}


void matrixMultiplication(double *A, double *B, double *C, int N){

    // declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);
        if (N*N > 512){
            threadsPerBlock.x = 512;
            threadsPerBlock.y = 512;
            blocksPerGrid.x = ceil(double(N)/double(threadsPerBlock.x));
            blocksPerGrid.y = ceil(double(N)/double(threadsPerBlock.y));
        }

    matrixMultiplicationKernel<<<blocksPerGrid,threadsPerBlock>>>(A, B, C, N);
}

void print(vector<double> h_C, int N){
    cout << "Result Matrix (GPU):" << endl;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            cout << h_C[i * N + j] << "\t";
        }
        cout << endl;
    }
}


int main()
{
    // Perform matrix multiplication C = A*B
    // where A, B and C are NxN matrices
    int N = 8;
    int SIZE = N*N;


    // Allocate memory on the host
    vector<double> h_A(SIZE);
    vector<double> h_B(SIZE);
    vector<double> h_C(SIZE);

    // Initialize matrices on the host
    for (int i=0; i<N; i++){
        for (int j=0; j<N; j++){
            h_A[i*N+j] = sin(i);
            h_B[i*N+j] = cos(j);
        }
    }

    // Allocate memory on the device
    dev_array<double> d_A(SIZE);
    dev_array<double> d_B(SIZE);
    dev_array<double> d_C(SIZE);

    d_A.set(&h_A[0], SIZE);
    d_B.set(&h_B[0], SIZE);

    matrixMultiplication(d_A.getData(), d_B.getData(), d_C.getData(), N);
    hipDeviceSynchronize();

    d_C.get(&h_C[0], SIZE);
    hipDeviceSynchronize();

    print(h_C,N);

    double *cpu_C;
    cpu_C=new double[SIZE];

    // Now do the matrix multiplication on the CPU
    double sum;
    for (int row=0; row<N; row++){
        for (int col=0; col<N; col++){
            sum = 0.f;
            for (int n=0; n<N; n++){
                sum += h_A[row*N+n]*h_B[n*N+col];
            }
            cpu_C[row*N+col] = sum;
        }
    }

    double err = 0;
    // Check the result and make sure it is correct
    for (int ROW=0; ROW < N; ROW++){
        for (int COL=0; COL < N; COL++){
            err += cpu_C[ROW * N + COL] - h_C[ROW * N + COL];
        }
    }

    cout << "Error: " << err << endl;

    return 0;
}